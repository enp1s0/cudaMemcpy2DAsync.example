#include <iostream>
#include <stdexcept>
#include <sstream>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK_ERROR(status) cuda_error_check(status, __FILE__, __LINE__, __func__)

inline void cuda_error_check(hipError_t error, const std::string filename, const std::size_t line, const std::string funcname, const std::string message = ""){
  if(error != hipSuccess){
    std::stringstream ss;
    ss<< hipGetErrorString( error );
    ss<<" ["<<filename<<":"<<line<<" in "<<funcname<<"]";
    throw std::runtime_error(ss.str());
  }
}

constexpr std::size_t A_M = 1lu << 5;
constexpr std::size_t A_N = 1lu << 4;

constexpr std::size_t B_M = 1lu << 2;
constexpr std::size_t B_N = 1lu << 2;

constexpr std::size_t B_OFFSET_M = 1lu << 2;
constexpr std::size_t B_OFFSET_N = 1lu << 2;

// Copy a part of matrix A as matrix B

// Col-major
//                     A_N
//     +--------------------------------------+
//     | A                                    |
//     |                                      |
//     | (B_OFFSET_M, B_OFFSET_N)             |
//     |           +------------------+       |
//     |           | B                |       |
//     |           |                  |       |
//     |           |                  |       |
// A_M |       B_M |                  |       |
//     |           |                  |       |
//     |           |                  |       |
//     |           +------------------+       |
//     |                   B_N                |
//     |                                      |
//     |                                      |
//     |                                      |
//     +--------------------------------------+

using data_t = float;

void print_matrix(
    const data_t* mat_ptr,
    const std::size_t ld,
    const std::size_t m,
    const std::size_t n
    ) {

  for (std::size_t i = 0; i < m; i++) {
    for (std::size_t j = 0; j < n; j++) {
      std::printf("%+.3e ", mat_ptr[i + j * ld]);
    }
    std::printf("\n");
  }
}

int main() {
  data_t* mat_A_ptr, *mat_B_ptr;
  CUDA_CHECK_ERROR(hipMallocManaged(&mat_A_ptr, sizeof(data_t) * A_M * A_N));
  CUDA_CHECK_ERROR(hipMallocManaged(&mat_B_ptr, sizeof(data_t) * B_M * B_N));

  for (std::size_t i = 0; i < A_M * A_N; i++) {
    mat_A_ptr[i] = i;
  }

  // Note: Consider that the matrices are stored in row-major
  CUDA_CHECK_ERROR(hipMemcpy2D(
      mat_B_ptr, sizeof(data_t) * B_M,
      mat_A_ptr + B_OFFSET_N * A_M + B_OFFSET_M, sizeof(data_t) * A_M,
      B_M * sizeof(data_t),
      B_N,
      hipMemcpyDefault
      ));

  std::printf("A_M = %lu\n", A_M);
  std::printf("A_N = %lu\n", A_N);
  std::printf("B_M = %lu\n", B_M);
  std::printf("B_N = %lu\n", B_N);
  std::printf("B_OFFSET_M = %lu\n", B_OFFSET_M);
  std::printf("B_OFFSET_N = %lu\n", B_OFFSET_M);

  std::printf("A\n");
  print_matrix(mat_A_ptr, A_M, A_M, A_N);
  std::printf("B\n");
  print_matrix(mat_B_ptr, B_M, B_M, B_N);
  std::printf("Correct Upper Left = %e\n", static_cast<double>(B_OFFSET_N * A_M + B_OFFSET_M));


  CUDA_CHECK_ERROR(hipFree(mat_A_ptr));
  CUDA_CHECK_ERROR(hipFree(mat_B_ptr));
}
